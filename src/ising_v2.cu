#include "hip/hip_runtime.h"
/*
**********************************************
*         - Ising model using CUDA -         *
*    GPU with multiple moments per thread    *
**********************************************
*/

#include "../inc/ising.h"

struct timeval startwtime, endwtime;
double p_time;

#define BLOCKSIZE 47

__global__ void kernel(int n, double* gpu_w, int* gpu_G, int* gpu_G_new, int grid_size)
{
	//! Each thread will calculate the value of multiple moments
	int moments_per_thread = BLOCKSIZE;

	//! Step for the next iteration
	int step = (n*n)/moments_per_thread;

	//! Thread ID
	int thread_id = blockIdx.x *blockDim.x + threadIdx.x;

	//! Variable to store the value of each moment
	double sum_value;

	//! The indices of the examined neighbors
	int idx_X, idx_Y;

	//! Moment's coordinates
	int mom_X = thread_id%grid_size;
	int mom_Y = thread_id/grid_size;

	if( thread_id < step )
	{
		int counter = 0;

		for(int i=thread_id; i < n*n; i+=step)
		{
			counter++;

			mom_X = i%n;
			mom_Y = i/n;

			if( (mom_X >= n) || (mom_Y >= n) )
			{
				printf(RED "Error 1: mom_X or mom_Y >= n\n" RESET_COLOR);
				break;
			}

			// if(is == 1)
			// 	printf("[%d,%d] %d\n", mom_X, mom_Y, i);

			sum_value = 0;

			//! Iterate through the moment's neighbors (k->X, l->Y axis)
		    for(int k=0; k<5; k++)
		        for(int l=0; l<5; l++)
		        {
		            //! Only edit the neighbors of the examined element
		            if((k == 2) && (l == 2))
		                continue;

		            //! Find the index of the examined neigbor
		            //! If the element is at a special position (i.e. a corner)
		            //! continue to the other side of the matrix
		            idx_X = (mom_X + (k-2) + n) % n;
		            idx_Y = (mom_Y + (l-2) + n) % n;

		            //! Calculate the new value
		            sum_value += gpu_w[l*5 + k] * gpu_G[idx_Y*n + idx_X];
		        }

		    //! If positive -> 1
		    //! If negative -> -1
		    if(sum_value > 1e-3)
		        gpu_G_new[mom_Y*n + mom_X] = 1;
		    else if(sum_value < -1e-3)
		        gpu_G_new[mom_Y*n + mom_X] = -1;
		    else
		        gpu_G_new[mom_Y*n + mom_X] = gpu_G[mom_Y*n + mom_X];
		}
		printf("iterations: %d\n", counter);
	}
	// else
	// 	printf(RED "Error 2: mom_X or mom_Y >= n\n" RESET_COLOR);
}

void ising(int *G, double *w, int k, int n)
{
    //! Store weights array to GPU
	double *gpu_w;
	hipMalloc(&gpu_w, 25*sizeof(double));
	hipMemcpy(gpu_w, w, 25*sizeof(double), hipMemcpyHostToDevice);

	//! Store G array to GPU
	int *gpu_G;
	hipMalloc(&gpu_G, n*n*sizeof(int));
	hipMemcpy(gpu_G, G, n*n*sizeof(int), hipMemcpyHostToDevice);

	//! GPU array to store the updated values
	int *gpu_G_new;
	hipMalloc(&gpu_G_new, n*n*sizeof(int));

	//! Temp pointer to swap gpu_G and gpu_G_new
	int *temp;

	//! Grid size comes from a combination of n and block size
	int blocks;
	int threads = BLOCKSIZE;
	if(n%threads == 0)
		blocks = n/threads;
	else
		blocks = n/threads + 1;

	printf("threads:%d\n", threads);
	printf("blocks:%d\n", blocks);
	printf("%d\n", blocks*blocks*threads);

	//! Define block and grid
	// dim3 dimBlock( threads, 1, 1 );
	// dim3 dimGrid ( blocks, blocks, 1 );

	//! Implement the process for k iterations
	for(int i = 0; i < k; i++)
	{
        kernel<<< blocks*blocks, threads >>>(n, gpu_w, gpu_G, gpu_G_new, blocks);

        //! Synchronize threads before swapping pointers
		hipDeviceSynchronize();

		//! Swap pointers for next iteration
		temp = gpu_G;
		gpu_G = gpu_G_new;
		gpu_G_new = temp;
	}

    //! Copy GPU final data to CPU memory
	hipMemcpy(G, gpu_G, n*n*sizeof(int), hipMemcpyDeviceToHost);

	//! Free allocated GPU memory
    hipFree(gpu_w);
	hipFree(gpu_G);
	hipFree(gpu_G_new);
}

int main(int argc, char *argv[])
{
	int n, k;

    if(argc > 1)
    {
        n = atoi(argv[1]);  // # number of elements (n*n)
        k = atoi(argv[2]);  // # iterations
    }else
    {
		n = 517;	// default value for n
        k = 1;		// default value for k
    }

	//! Array that will keep the init binary file info
	int *G = (int*)malloc(n*n * sizeof(int));

	//! Weights
    double weights[] = {0.004, 0.016, 0.026, 0.016, 0.004,
                		0.016, 0.071, 0.117, 0.071, 0.016,
            			0.026, 0.117, 0    , 0.117, 0.026,
            			0.016, 0.071, 0.117, 0.071, 0.016,
            			0.004, 0.016, 0.026, 0.016, 0.004};

	//! Open binary file and write contents to G array
    FILE *fptr = fopen("conf-files/conf-init.bin","rb");
    if (fptr == NULL)
	{
        printf("Error opening file");
        exit(1);
    }
    fread(G, sizeof(int), n*n, fptr);
	fclose(fptr);

	//! ========= START POINT =========
    gettimeofday (&startwtime, NULL);

    //! Implement ising procedure
    ising(G, weights, k, n);

	//! ========= END POINT =========
    gettimeofday (&endwtime, NULL);
    p_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
  		      + endwtime.tv_sec - startwtime.tv_sec);

	//! Name of conf file depending on k value
	char filename[25];
	snprintf(filename, sizeof(filename), "conf-files/conf-%d.bin", k);

	//! Compare updated data with the correct data (for k = 1, 4, 11)
	int *data = (int*)malloc(n*n * sizeof(int));
	int isWrong = 0;

	fptr = fopen(filename,"rb");
	fread(data, sizeof(int), n*n, fptr);
	fclose(fptr);
	for(int i = 0; i < n*n; i++)
		if(data[i] != G[i])
			isWrong = 1;

	//! Check if any comparison failed
	if (!isWrong)
		printf("[k=%d]" GREEN " CORRECT\n" RESET_COLOR, k);
	else
		printf("[k=%d]" RED " WRONG\n" RESET_COLOR, k);

	printf(RED "Real Time: %f\n", p_time);

	//! Free allocated GPU memory
    free(G);
	free(data);

    return 0;
}
