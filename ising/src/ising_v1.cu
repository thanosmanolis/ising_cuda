#include "hip/hip_runtime.h"
/*
****************************************
*      - Ising model using CUDA -      *
*    GPU with one thread per moment    *
****************************************
*/

#include "../inc/hip/hip_runtime.h"

#define BLOCKSIZE 256

__global__ void kernel(int n,  double* gpu_w, int* gpu_G, int* gpu_G_new)
{
	//! Thread ID (from 0 to n*n)
	int thread_id = blockIdx.x*blockDim.x + threadIdx.x;

	//! Variable to store the value of each moment
	double sum_value;

	//! Moment's coordinates
	int mom_X = thread_id%n;
	int mom_Y = thread_id/n;

	//! The indices of the examined neighbors
	int idx_X, idx_Y;

	if( thread_id < n*n )
	{
		//! Iterate through the moment's neighbors (k->X, l->Y axis)
	    for(int k=0; k<5; k++)
	        for(int l=0; l<5; l++)
	        {
	            //! Only edit the neighbors of the examined element
	            if((k == 2) && (l == 2))
	                continue;

	            //! Find the index of the examined neigbor
	            //! If the element is at a special position (i.e. a corner)
	            //! continue to the other side of the matrix
	            idx_X = (mom_X + (k-2) + n) % n;
	            idx_Y = (mom_Y + (l-2) + n) % n;

	            //! Calculate the new value
	            sum_value += gpu_w[l*5 + k] * gpu_G[idx_Y*n + idx_X];
	        }

	    //! If positive -> 1
	    //! If negative -> -1
	    if(sum_value > 1e-3)
	        gpu_G_new[mom_Y * n + mom_X] = 1;
	    else if(sum_value < -1e-3)
	        gpu_G_new[mom_Y * n + mom_X] = -1;
	    else
	        gpu_G_new[mom_Y * n + mom_X] = gpu_G[mom_Y * n + mom_X];
	}
}

void ising(int *G, double *w, int k, int n)
{
    //! Store weights array to GPU
	double *gpu_w;
	hipMalloc(&gpu_w, 25*sizeof(double));
	hipMemcpy(gpu_w, w, 25*sizeof(double), hipMemcpyHostToDevice);

	//! Store G array to GPU
	int *gpu_G;
	hipMalloc(&gpu_G, n*n*sizeof(int));
	hipMemcpy(gpu_G, G, n*n*sizeof(int), hipMemcpyHostToDevice);

	//! GPU array to store the updated values
	int *gpu_G_new;
	hipMalloc(&gpu_G_new, n*n*sizeof(int));

	//! Temp pointer to swap gpu_G and gpu_G_new
	int *temp;
	
	//! Grid size comes from a combination of n and BLOCKSIZE
	int threads = BLOCKSIZE;
	int blocks;
	if(n*n%threads == 0)
		blocks = n*n/threads;
	else
		blocks = n*n/threads + 1;

	printf(" threads per block: %d\n", threads);
	printf("            blocks: %d\n", blocks);
	printf("  threads in total: %d\n", blocks*threads);

	//! Implement the process for k iterations
	for(int i = 0; i < k; i++)
	{
        kernel<<< blocks , threads >>>(n, gpu_w, gpu_G, gpu_G_new);

        //! Synchronize threads before swapping pointers
		hipDeviceSynchronize();

		//! Swap pointers for next iteration
		temp = gpu_G;
		gpu_G = gpu_G_new;
		gpu_G_new = temp;
	}

    //! Copy GPU final data to CPU memory
	hipMemcpy(G, gpu_G, n*n*sizeof(int), hipMemcpyDeviceToHost);

	//! Free allocated GPU memory
    hipFree(gpu_w);
	hipFree(gpu_G);
	hipFree(gpu_G_new);
}
